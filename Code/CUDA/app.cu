#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void funzione()
{
  //printf("\t\t\t\t\t[DEVICE] Hello World!\n");
  
	int thrId = threadIdx.x;
	int blkId = blockIdx.x;
	int thrNum = blockDim.x;
  int blkNum = gridDim.x;
  printf("\t\t\t\t\t[DEVICE] Hello World! I am thread #%d out of %d, and I belong to block #%d out of %d\n", thrId, thrNum, blkId, blkNum);
  
  return;
}

int main()
{
  printf("[HOST] Hello World!\n");
  
  funzione<<<3,5>>>();

  printf("[HOST] Some more work on host\n");

  hipDeviceSynchronize();
  
  printf("[HOST] Device ended its work!\n");
  
  return 0;
}
